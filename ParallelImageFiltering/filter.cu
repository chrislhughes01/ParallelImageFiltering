#include "hip/hip_runtime.h"
#include "filter.cuh"
#include <hip/hip_runtime.h>
#include <>

#define FILTER_SIZE 7 // 7 x 7 Kernel
#define BLOCK_SIZE 16

__constant__ int filter[FILTER_SIZE][FILTER_SIZE] = {
		{1, 2, 3, 4, 3, 2, 1},
		{2, 6, 8, 10, 8, 6, 2},
		{3, 8, 12, 15, 12, 8, 3},
		{4, 10, 15, 20, 15, 10, 4},
		{3, 8, 12, 15, 12, 8, 3},
		{2, 6, 8, 10, 8, 6, 2},
		{1, 2, 3, 4, 3, 2, 1}
};

__device__ int clamp(int value, int minVal, int maxVal) {
	return value < minVal ? minVal : (value > maxVal ? maxVal : value);
}

__global__ void imageFilterKernel(unsigned char* input, unsigned char* output, int width, int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height) return;

	int filterSum = 316;

	int pixelSum = 0;
	for (int i = -3; i <= 3; i++) {
		for (int j = -3; j <= 3; j++) {
			int nx = x + i;
			int ny = y + j;
			if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
				pixelSum += input[ny * width + nx] * filter[i + 3][j + 3];
			}
		}
	}

	int blurredPixel = (int) ((float) pixelSum / (float)filterSum + 0.5f);
	output[y * width + x] = clamp(blurredPixel, 0, 255);
}

void applyCUDAFilter(unsigned char* input, unsigned char* output, int width, int height) {
	unsigned char* d_input, * d_output, *d_temp;
	hipStream_t stream;
	hipStreamCreate(&stream);
	size_t imgSize = width * height * sizeof(unsigned char);

	hipMalloc((void**)&d_input, imgSize);
	hipMalloc((void**)&d_output, imgSize);
	hipMalloc((void**)&d_temp, imgSize);
	hipMemcpyAsync(d_input, input, imgSize, hipMemcpyHostToDevice, stream);

	dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

	for (int i = 0; i < 5; i++) {
		imageFilterKernel << <gridSize, blockSize >> > (d_input, d_output, width, height);
		hipMemcpyAsync(d_temp, d_output, imgSize, hipMemcpyDeviceToDevice, stream);
		hipMemcpyAsync(d_input, d_input, imgSize, hipMemcpyDeviceToDevice, stream);
	}

	hipMemcpyAsync(output, d_output, imgSize, hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	hipFree(d_input);
	hipFree(d_output);
	hipFree(d_temp);
}